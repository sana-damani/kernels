#include "hip/hip_runtime.h"
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 1; 
 
__global__ 
void relu(int* d_in) 
{
	// map function: f(x) = x if x >= 0, 0 otherwise
	int val = d_in[threadIdx.x];
	d_out[threadIdx.x] = val < 0 ? 0 : val;
}
 
int main()
{
 
	int *h_in;
	const int size = N*sizeof(int);
 
	malloc((void**)&h_in, size); 
	malloc((void**)&h_out, size);  

	for (int ii = 0; ii < N; ii++) {
	    if (ii % 2)
		h_in[ii] = ii;
	    else
		h_in[ii] = -ii;
	}
	hipMalloc((void**)&d_in, size); 
	hipMalloc((void**)&d_out, size); 
	hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice); 
	
	dim3 dimBlock(blocksize, N);
	dim3 dimGrid(1, 1);
	hello<<<dimGrid, dimBlock>>>(d_in);
	hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost); 
	for (int ii = 0; ii < N; ii++)
	    printf("\nin[%d]=%d\tout[%d]=%d", ii, h_in[ii], ii, h_out[ii]);
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);
	
	return EXIT_SUCCESS;
}
