
#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16; 
 
__global__ 
void relu(int* d_in, int *d_out) 
{
	// map function: f(x) = x if x >= 0, 0 otherwise
	int val = d_in[threadIdx.x];
	d_out[threadIdx.x] = val < 0 ? 0 : val;
}
 
int main()
{
 
	int *h_in, *h_out, *d_in, *d_out;
	const int size = N*sizeof(int);
 
	h_in = (int*)malloc(size); 
	h_out = (int*)malloc(size); 

	for (int ii = 0; ii < N; ii++) {
	    if (ii % 2)
		h_in[ii] = ii;
	    else
		h_in[ii] = -ii;
	}
	hipMalloc((void**)&d_in, size); 
	hipMalloc((void**)&d_out, size); 
	hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice); 
	
	dim3 dimBlock(N, 1);
	dim3 dimGrid(1, 1);
	relu<<<dimGrid, dimBlock>>>(d_in, d_out);
	hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost); 
	for (int ii = 0; ii < N; ii++)
	    printf("\nin[%d]=%d\tout[%d]=%d", ii, h_in[ii], ii, h_out[ii]);
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);
	
	return EXIT_SUCCESS;
}
